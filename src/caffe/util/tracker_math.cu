#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>
#include <cstdlib>
#include <cstring>

#include "caffe/common.hpp"
#include "caffe/util/tracker_math.hpp"
#include "hipblas.h"
#define THREADS_PER_BLOCK_CSR 32

namespace caffe {

template <typename Dtype>
__global__ void toInt_kernel(int n, const Dtype* in, int* out)
{
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = static_cast<int>(in[index]);
  }
}

template <typename Dtype>
void tracker_gpu_toInt(int n, const Dtype* in, int* out)
{
  toInt_kernel<Dtype><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(n, in, out);
}

template void tracker_gpu_toInt<float>(int n, const float* in, int* out);
template void tracker_gpu_toInt<double>(int n, const double* in, int* out);



template <typename Dtype>
__global__ void toDtype_kernel(int n, const int* in, Dtype* out)
{
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = static_cast<Dtype>(in[index]);
  }
}

template <typename Dtype>
void tracker_gpu_toDtype(int n, const int* in, Dtype* out)
{
  toDtype_kernel<Dtype><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(n, in, out);
}

template void tracker_gpu_toDtype<float>(int n, const int* in, float* out);
template void tracker_gpu_toDtype<double>(int n, const int* in, double* out);

template <>
void tracker_gpu_csr_gemm_cusparse<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, int nzz, const float* A, const int* indices, const int* ptr, const float* B, const float beta,
    float* C, const CBLAS_ORDER orderC) {

  //std::cout << "M: " << M << " N: " << N << " K: " << K << " NZZ: " << nzz <<"\n"  ;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipsparseOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPSPARSE_OPERATION_NON_TRANSPOSE : HIPSPARSE_OPERATION_TRANSPOSE;
  hipsparseOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;

  float* A_t;
  int* A_t_indices;
  int* A_t_ptr;
  
  int msparse = (TransA == CblasNoTrans) ? M : K;
  int ksparse = (TransA == CblasNoTrans) ? K : M;
  
  bool reuiqre_transpose_A = (cuTransA == HIPSPARSE_OPERATION_TRANSPOSE) && (cuTransB == HIPSPARSE_OPERATION_TRANSPOSE);
  //LOG(ERROR) << "Require Transpose A? " << reuiqre_transpose_A;
  if (reuiqre_transpose_A){
    CUDA_CHECK(hipMalloc((void**)&A_t, sizeof(float)*nzz));
    CUDA_CHECK(hipMalloc((int**)&A_t_indices, sizeof(int)*nzz));
    CUDA_CHECK(hipMalloc((int**)&A_t_ptr, sizeof(int)*(ksparse+1)));
    CUSPARSE_CHECK(hipsparseScsr2csc(Caffe::cusparse_handle(), msparse, ksparse, nzz, A, ptr, indices, A_t, A_t_indices, A_t_ptr, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO));
  }
  
  if (orderC == CblasRowMajor){
    float* Ct;
    CUDA_CHECK(hipMalloc((void**)&Ct, sizeof(float)*M*N));
    const float zero = 0.0;
    const float one = 1.0;
    if (reuiqre_transpose_A){
      CUSPARSE_CHECK(hipsparseScsrmm2(Caffe::cusparse_handle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, cuTransB, ksparse, N, msparse,nzz, &alpha, Caffe::cusparse_mat_descr(), A_t, A_t_ptr, A_t_indices, B,  ldb, &zero, Ct, M));
      CUDA_CHECK(hipFree(A_t));
      CUDA_CHECK(hipFree(A_t_indices));
      CUDA_CHECK(hipFree(A_t_ptr));
    }else{
      CUSPARSE_CHECK(hipsparseScsrmm2(Caffe::cusparse_handle(), cuTransA, cuTransB, msparse, N, ksparse,nzz, &alpha, Caffe::cusparse_mat_descr(), A, ptr, indices, B,  ldb, &zero, Ct, M));
    }
    CUBLAS_CHECK(hipblasSgeam(Caffe::cublas_handle(), HIPBLAS_OP_T , HIPBLAS_OP_N, N, M, &one, Ct, M, &beta, C, N, C, N));
    CUDA_CHECK(hipFree(Ct));
  }else{
      
    //this is the default of CUSPARSE by the Matrix B is by default rowmajor
    if (reuiqre_transpose_A){
      CUSPARSE_CHECK(hipsparseScsrmm2(Caffe::cusparse_handle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, cuTransB, ksparse, N, msparse,nzz, &alpha, Caffe::cusparse_mat_descr(), A_t, A_t_ptr, A_t_indices, B,  ldb, &beta, C, M));      
      CUDA_CHECK(hipFree(A_t));
      CUDA_CHECK(hipFree(A_t_indices));
      CUDA_CHECK(hipFree(A_t_ptr));
    }else{
      //LOG(ERROR) << "HERE!!!! " << (cuTransA == HIPSPARSE_OPERATION_TRANSPOSE) << ", " << (cuTransB == HIPSPARSE_OPERATION_TRANSPOSE) << ", " << msparse << ", " << N << ", " << ksparse << ", " << nzz << ", " << ldb << ", " << M;
      CUSPARSE_CHECK(hipsparseScsrmm2(Caffe::cusparse_handle(), cuTransA, cuTransB, msparse, N, ksparse,nzz, &alpha, Caffe::cusparse_mat_descr(), A, ptr, indices, B,  ldb, &beta, C, M));
    }
  }
}


template <>
void tracker_gpu_csr_gemm_cusparse<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, int nzz, const double* A, const int* indices, const int* ptr, const double* B, const double beta,
    double* C, const CBLAS_ORDER orderC) {

  //std::cout << "M: " << M << "N: " << N << "K: " << K << "NZZ: " << nzz  ;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipsparseOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPSPARSE_OPERATION_NON_TRANSPOSE : HIPSPARSE_OPERATION_TRANSPOSE;
  hipsparseOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;

  double* A_t;
  int* A_t_indices;
  int* A_t_ptr;
  
  int msparse = (TransA == CblasNoTrans) ? M : K;
  int ksparse = (TransA == CblasNoTrans) ? K : M;
  
  bool reuiqre_transpose_A = (cuTransA == HIPSPARSE_OPERATION_TRANSPOSE) && (cuTransB == HIPSPARSE_OPERATION_TRANSPOSE);
  if (reuiqre_transpose_A){
    CUDA_CHECK(hipMalloc((void**)&A_t, sizeof(double)*nzz));
    CUDA_CHECK(hipMalloc((int**)&A_t_indices, sizeof(int)*nzz));
    CUDA_CHECK(hipMalloc((int**)&A_t_ptr, sizeof(int)*(ksparse+1)));
    CUSPARSE_CHECK(hipsparseDcsr2csc(Caffe::cusparse_handle(), msparse, ksparse, nzz, A, ptr, indices, A_t, A_t_indices, A_t_ptr, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO));
  }

  
  if (orderC == CblasRowMajor){
    double* Ct;
    CUDA_CHECK(hipMalloc((void**)&Ct, sizeof(double)*M*N));
    const double zero = 0.0;
    const double one = 1.0;
    if (reuiqre_transpose_A){
      CUSPARSE_CHECK(hipsparseDcsrmm2(Caffe::cusparse_handle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, cuTransB, ksparse, N, msparse,nzz, &alpha, Caffe::cusparse_mat_descr(), A_t, A_t_ptr, A_t_indices, B,  ldb, &zero, Ct, M));
      CUDA_CHECK(hipFree(A_t));
      CUDA_CHECK(hipFree(A_t_indices));
      CUDA_CHECK(hipFree(A_t_ptr));
    }else{
      CUSPARSE_CHECK(hipsparseDcsrmm2(Caffe::cusparse_handle(), cuTransA, cuTransB, msparse, N, ksparse,nzz, &alpha, Caffe::cusparse_mat_descr(), A, ptr, indices, B,  ldb, &zero, Ct, M));
    }
    CUBLAS_CHECK(hipblasDgeam(Caffe::cublas_handle(), HIPBLAS_OP_T , HIPBLAS_OP_N, N, M, &one, Ct, M, &beta, C, N, C, N));
    CUDA_CHECK(hipFree(Ct));
  }else{
    //this is the default of CUSPARSE by the Matrix B is by default rowmajor
    if (reuiqre_transpose_A){
      CUSPARSE_CHECK(hipsparseDcsrmm2(Caffe::cusparse_handle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, cuTransB, ksparse, N, msparse,nzz, &alpha, Caffe::cusparse_mat_descr(), A_t, A_t_ptr, A_t_indices, B,  ldb, &beta, C, M));      
      CUDA_CHECK(hipFree(A_t));
      CUDA_CHECK(hipFree(A_t_indices));
      CUDA_CHECK(hipFree(A_t_ptr));
    }else{
      CUSPARSE_CHECK(hipsparseDcsrmm2(Caffe::cusparse_handle(), cuTransA, cuTransB, msparse, N, ksparse,nzz, &alpha, Caffe::cusparse_mat_descr(), A, ptr, indices, B,  ldb, &beta, C, M));
    }
  }
}


}  // namespace caffe
